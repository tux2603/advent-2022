
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

#define DEBUG
// #define TEST_DATA

#ifdef TEST_DATA
#    define NUM_SENSORS 14
#    define INPUT_FILE "cleaned_input_test.txt"
#    define MAX_SEARCH 20
#else
#    define NUM_SENSORS 38
#    define INPUT_FILE "cleaned_input.txt"
#    define MAX_SEARCH 4000000
#endif

// Found using the cuda occupancy calculator
#define THREADS_PER_BLOCK 768
#define BLOCKS_PER_SM 2
#define ITERATIONS_PER_KERNEL 40

#define MIN(a, b) (((a) < (b)) ? (a) : (b))

typedef struct {
    int32_t x;
    int32_t y;
    int32_t range;
} sensor_t;

__global__ void search_kernel(sensor_t *sensors, int32_t *out_x, int32_t *out_y, bool *found, int start_y, int end_y) {
    __shared__ sensor_t shared_sensors[NUM_SENSORS];

    // Copy the sensor data into shared memory since we're going to be using it so much
    if (threadIdx.x == 0) {
        for (int i = 0; i < NUM_SENSORS; i++) {
            shared_sensors[i] = sensors[i];
        }
    }

    // Resync all the threads now that the data is copied
    __syncthreads();

    for (int32_t y = start_y + blockIdx.x; y < end_y; y += gridDim.x) {
        for (int32_t x = threadIdx.x; x <= MAX_SEARCH; x += blockDim.x) {
            bool in_range = false;

            for (uint8_t i = 0; i < NUM_SENSORS; i++) {
                int32_t distance = abs(shared_sensors[i].x - x) + abs(shared_sensors[i].y - y);

                in_range |= (distance <= shared_sensors[i].range);
            }

            if (!in_range) {
                *out_x = x;
                *out_y = y;
                *found = true;
            }
        }
    }
}

/**
 * @brief Reads the raw sensor data from a precleaned input file
 * 
 * @param fp A file pointer already opened and pointing at the input file
 * @param sensors A pointer to a shared array that will hold the sensor data
 */
void parse_sensor_data(FILE *fp, sensor_t *sensors) {
    char *buffer = (char * ) malloc(34 * sizeof(char));
    size_t length = 34;

    // Parse the sensor data from the file
    for (int i = 0; i < NUM_SENSORS; i++) {
        // Read a line from the file
        if (!fgets(buffer, length, fp)) {
            printf("Error reading file\n");
            return;
        }

        else {
            int32_t beacon_x, beacon_y;

            // Parse the sensor data
            sscanf(buffer, "%d %d %d %d", &sensors[i].x, &sensors[i].y, &beacon_x, &beacon_y);

            // Calculate the range of the sensor
            sensors[i].range = abs(beacon_x - sensors[i].x) + abs(beacon_y - sensors[i].y);
        }
    }

    // Free the buffer
    if (buffer) {
        free(buffer);
    }
}


int main(int argc, char **argv) {

    // Create two 32 bit unsigned integers to copy back beacon location into, and a boolean flag to indicate if the beacon was found
    int32_t *x, *y;
    bool *found;
    hipMallocManaged(&x, sizeof(int32_t));
    hipMallocManaged(&y, sizeof(int32_t));
    hipMallocManaged(&found, sizeof(bool));
    *x = 0;
    *y = 0;
    *found = false;

    // Create a shared array to store the sensor data
    sensor_t *sensors;
    hipMallocManaged(&sensors, NUM_SENSORS * sizeof(sensor_t));


    // Open the file "cleaned_input.txt" for parsing
    FILE *fp = fopen(INPUT_FILE, "r");

    // Parse the sensor data from the file
    parse_sensor_data(fp, sensors);

    // Close the file
    fclose(fp);

#ifdef DEBUG
    // Print out all of the sensors for debugging
    for (int i = 0; i < NUM_SENSORS; i++) {
        printf("Sensor %d is at (%d, %d) with range of %d\n", i + 1, sensors[i].x, sensors[i].y, sensors[i].range);
    }
#endif

    // Get some information on the cuda device that this is running on
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // I'm going to be lazy and assume a device with compute capability of 8.6 when doing the math to figure out
    //  how many threads to use per block and how many blocks to use per grid.
    int major_version = prop.major;
    int minor_version = prop.minor;
    int sm_count = prop.multiProcessorCount;
    int max_concurrent_blocks = sm_count * BLOCKS_PER_SM;

    if (major_version != 8 || minor_version != 6) {
        printf("\033[1;31mWARNING!\033[0m This program was optimized for cuda devices with compute capability 8.6, but this device has compute capability %d.%d\n", major_version, minor_version);
        printf("         The program will probably still run, it just might be a bit slower than it would be otherwise because of reduced occupancy\n");
    }

#ifdef DEBUG
    printf("There are %d SMs on this device, this will allow %d thread blocks to run simultaneously\n", sm_count, max_concurrent_blocks);
#endif

#ifdef TEST_DATA
    search_kernel<<<MAX_SEARCH, MAX_SEARCH>>>(sensors, x, y, found, 0, MAX_SEARCH);
    cudaDeviceSynchronize();
#else
#    ifdef DEBUG
    clock_t start = clock();
#    endif

    // Search for the beacon
    for (int i = 0; i <= MAX_SEARCH; i += max_concurrent_blocks * ITERATIONS_PER_KERNEL) {
        int32_t end_y = MIN(MAX_SEARCH, i + max_concurrent_blocks * ITERATIONS_PER_KERNEL);
        search_kernel<<<max_concurrent_blocks, THREADS_PER_BLOCK>>>(sensors, x, y, found, i, end_y);

        // Wait for the kernel to finish
        hipDeviceSynchronize();

#    ifdef DEBUG
        clock_t elapsed = clock();
        printf("Searching for beacon... %d / %d lines searched (%d%%)", i, MAX_SEARCH, (int)((float)i / (float)MAX_SEARCH * 100.0f));
        float seconds_elapsed = (float)(elapsed - start) / (float)CLOCKS_PER_SEC;
        float estimated_total_time = seconds_elapsed * ((float)MAX_SEARCH / (float)i);
        float estimated_time_remaining = estimated_total_time - seconds_elapsed;
        printf(" (%.2f seconds elapsed, ETA %.2f seconds)\n", seconds_elapsed, estimated_time_remaining);
#    endif

        // Check if the beacon was found
        if (*found) {
            break;
        }
    }
#endif

    // Print out the beacon location
    if (*found) {
        printf("Beacon found at (%d, %d)\n", *x, *y);
        uint64_t frequency = (uint64_t)*x * 4000000 + (uint64_t)*y;
        printf("Frequency is %lu\n", frequency);
    }

    else {
        printf("Beacon not found\n");
    }
    return 0;
}